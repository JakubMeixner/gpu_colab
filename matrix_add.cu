
// This program computes a simple version of matrix multiplication
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

__global__ void matrixMul(const int *a, const int *b, int *c, int N) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterate over row, and down column
  c[row * N + col] = 0;
  for (int k = 0; k < N; k++) {
    // Accumulate results for a single element
    c[row * N + col] += a[row * N + k] * b[k * N + col];
  }
}

// Check result on the CPU
void verify_result(vector<int> &a, vector<int> &b, vector<int> &c, int N) {
  for (int row = 0; row < N; row++) {
    for (int col = 0; col < N; col++) {
      int tmp = 0; // For every element in the row-column pair
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += a[row * N + k] * b[k * N + col];
      }
      // Check against the CPU result
      assert(tmp == c[row * N + col]);
    }
  }
}

int main() {
  int N = 1 << 10;  // Matrix size of 1024 x 1024;

  // Size (in bytes) of matrix
  size_t bytes = N * N * sizeof(int);

  // Host vectors
  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);

  // Initialize matrices
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  // Allocate device memory
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = N / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

  // Copy back to the host
  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  // Check result
  verify_result(h_a, h_b, h_c, N);

  cout << "COMPLETED SUCCESSFULLY\n";

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
